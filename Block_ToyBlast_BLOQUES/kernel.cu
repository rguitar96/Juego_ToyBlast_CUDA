#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <time.h>
#include <stdio.h>
#include <conio.h>
#include <iostream>
#include <sstream>
#include <sys/stat.h>
#include <vector>
#include <Windows.h>
#include <fstream>

using namespace std;

//FUNCIONES CPU
bool existeFichero(const string& fichero);
void setFontSize(int FontSize);
void cargarPartida(const string& fichero, int tam_tesela);
void tableroAleatorio(vector<vector<int>>& tablero, int dificultad);
void nuevaPartida(vector<vector<int>>& tablero, int dificultad, int filas, int columnas, int puntuacion, int tam_tesela);
void imprimirTablero(vector<vector<int>>& tablero);
void guardarPartida(vector<vector<int>> tablero, string nombre, int filas, int columnas, int dificultad, int puntuacion);
bool quedanMovimientosF(vector<vector<int>> tablero);

#define TILE_WIDTH 16

//FUNCIONES GPU
__global__ void ToyBlast(int *tablero, int filas, int columnas, int fila, int columna, int *puntuacion);
__device__ void eliminarPieza(int *tablero, int filas, int columnas, int fila, int columna, int valor_ini, int *cont);
__device__ void bombaRotorH(int* tablero, int filas, int columnas, int fila, int columna);
__device__ void bombaRotorV(int* tablero, int filas, int columnas, int fila, int columna);
__device__ void bombaTNT(int* tablero, int filas, int columnas, int fila, int columna);
__device__ void bombaPuzzle(int* tablero, int filas, int columnas, int fila, int columna);

int main(int argc, char *argv[])
{
	SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
	hipDeviceProp_t propiedades;
	hipGetDeviceProperties(&propiedades, 0);
	int hilos_max = propiedades.maxThreadsPerBlock;
	int tam_tesela = TILE_WIDTH;

	//SI HAY M�S ARGUMENTOS QUE argv[0]
	if (argc > 1) {
		//SI S�LO HAY UN ARGUMENTO M�S, SE CARGA EL FICHERO CON ESE NOMBRE
		if (argc == 2) {
			cargarPartida(argv[1], tam_tesela);
		}
		else {
			//SI HAY TRES ARGUMENTOS M�S, SE CARGA LA PARTIDA CON (dificultad, filas, columnas)
			if (argc == 4) {
				int dificultad = atoi(argv[1]);
				int filas = atoi(argv[2]);
				int columnas = atoi(argv[3]);
				vector<vector<int>> tablero;
				tablero.resize(filas, vector<int>(columnas, 0));

				//SE ALEATORIZA EL TABLERO Y SE INICIA LA PARTIDA
				tableroAleatorio(tablero, dificultad);
				nuevaPartida(tablero, dificultad, filas, columnas, 0, tam_tesela);

			}
			else {
				cout << "El archivo debe ejecutarse de una de las tres maneras:\n-Sin argumentos.\n-Con un unico argumento indicando el nombre del fichero a cargar.\n-Con tres argumentos indicando dificultad, filas y columnas del nuevo tablero.\n";
			}
		}
	}
	else {

		bool valido = false;
		bool nueva = true;
		string fichero;

		while (!valido) {
			cout << "Si desea cargar una partida, introduzca su nombre con la extension (.txt). Presione enter para iniciar una nueva partida.\n";

			getline(cin, fichero);

			if (fichero != "") {
				//COMPROBAMOS SI EL FICHERO EXISTE, SI NO VOLVEMOS A PREGUNTAR
				if (existeFichero(fichero)) {
					valido = true;
					nueva = false;
				}
				else {
					cout << "El fichero no existe.";
				}
			}
			else {
				//NUEVA PARTIDA
				valido = true;
			}
		}

		if (nueva) {
			//INICIO NUEVA PARTIDA

			cout << "Por favor, introduzca el numero de filas.\n";
			int filas;
			
			//GET FILAS
			while (!(cin >> filas)) {
				cin.clear();
				cin.ignore((std::numeric_limits<std::streamsize>::max)(), '\n');
				cerr << "Por favor, introduzca un numero de fila valido.\n";
			}

			cout << "Por favor, introduzca el numero de columnas.\n";
			int columnas;

			//GET COLUMNAS
			while (!(cin >> columnas)) {
				cin.clear();
				cin.ignore((std::numeric_limits<std::streamsize>::max)(), '\n');
				cerr << "Por favor, introduzca un numero de columna valido.\n";
			}

			//GET DIFICULTAD
			int dificultad = -1;
			while (dificultad != 2 && dificultad != 1) {
				cout << "Por favor, introduzca la dificultad (1 para dificultad facil y 2 para dificil).\n";
				while (!(cin >> dificultad)) {
					cin.clear();
					cin.ignore((std::numeric_limits<std::streamsize>::max)(), '\n');
					cerr << "Por favor, introduzca un numero de dificultad valido.\n";
				}
				if (dificultad != 2 && dificultad != 1) {
					cout << "Entrada invalida.\n";
				}
			}
			
			//SI EL TABLERO NO CABE EN PANTALLA, SE HACE LA LETRA M�S PEQUE�A
			if (columnas > 48) {
				setFontSize(11);
				if (columnas > 55) setFontSize(8);
				if (columnas > 75) setFontSize(6);
				if (columnas > 90) setFontSize(4);
			}

			vector<vector<int>> tablero;
			tablero.resize(filas, vector<int>(columnas, 0));

			tableroAleatorio(tablero, dificultad);

			nuevaPartida(tablero, dificultad, filas, columnas, 0, tam_tesela);
		}
		else {
			cargarPartida(fichero, tam_tesela);
		}
	}
	return 0;
}

inline bool existeFichero(const string& fichero) {
	struct stat buffer;
	return (stat(fichero.c_str(), &buffer) == 0);
}

void cargarPartida(const string& fichero, int tam_tesela) {
	//YA SE HA COMPROBADO QUE EL ARCHIVO EXISTE, CARGAR ARCHIVO EXISTENTE
	vector<int> datavec;

	ifstream infile;
	infile.open(fichero, ios::in | ios::binary);

	while (infile) {
		int val;
		infile.read(reinterpret_cast<char *>(&val), sizeof(int));
		if (infile.bad()) {
			throw std::runtime_error("Failed to read from infile!");
		}
		if (infile.eof()) break;
		datavec.push_back(val);
	}

	//FORMATO DEL FICHERO: un vector de enteros con vector[0]=puntuacion, vector[1]=filas, vector[2]=columnas, vector[3]=dificultad, y la matriz en una lista unidimensional de enteros
	int puntuacion = datavec[0];
	datavec.erase(datavec.begin());
	int filas = datavec[0];
	datavec.erase(datavec.begin());
	int dificultad = datavec[0];
	datavec.erase(datavec.begin());

	int columnas = (datavec.size()) / filas;

	vector<vector<int>> tablero;
	tablero.resize(filas, vector<int>(columnas, 0));
	imprimirTablero(tablero);
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			tablero[i][j] = datavec[0];
			datavec.erase(datavec.begin());
		}
	}
	nuevaPartida(tablero, dificultad, filas, columnas, puntuacion, tam_tesela);

}

void setFontSize(int FontSize)
{
	//PONER LA FUENTE A CIERTO TAMA�O
	CONSOLE_FONT_INFOEX info = { 0 };
	info.cbSize = sizeof(info);
	info.dwFontSize.Y = FontSize;
	info.FontWeight = FW_NORMAL;
	wcscpy(info.FaceName, L"Lucida Console");
	SetCurrentConsoleFontEx(GetStdHandle(STD_OUTPUT_HANDLE), NULL, &info);
}

void tableroAleatorio(vector<vector<int>>& tablero, int dificultad) {
	srand(time(NULL));
	//MODIFICA TODOS LOS HUECOS LIBRES DEL TABLERO (QUE SEAN 0) CON BLOQUES ALEATORIOS
	if (dificultad == 1) {
		for (int i = 0; i < tablero.size(); ++i) {
			for (int j = 0; j < tablero[0].size(); ++j) {
				if (tablero[i][j] == 0) tablero[i][j] = rand() % 5 + 1;
			}
		}
	}
	else {
		for (int i = 0; i < tablero.size(); ++i) {
			for (int j = 0; j < tablero[0].size(); ++j) {
				if (tablero[i][j] == 0) tablero[i][j] = rand() % 6 + 1;
			}
		}
	}
}

void imprimirTablero(vector<vector<int>>& tablero) {
	//IMPRIMIR CABECERA
	cout << "#_____________TABLERO_DE_JUEGO_____________\n\n       ";
	SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 8 + 15 * 16);
	for (int i = 0; i < tablero[0].size(); ++i) {
		if (i % 2 == 0) {
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 1 + 16 * 8);
		}
		else {
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15 + 16 * 1);
		}
		cout << " ";
		if (i + 1 < 10) cout << " ";
		cout << i + 1;
		if (i + 1 < 100) cout << " ";
	}
	cout << "\n";

	//IMPRIMIR CADA POSICI�N
	for (int i = 0; i < tablero.size(); ++i) {
		if (i % 2 == 0) {
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 1 + 16 * 8);
		}
		else {
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15 + 16 * 1);
		}
		cout << "  ";
		if (i + 1<10) cout << " ";
		if (i + 1 < 100) cout << " ";
		cout << i + 1;
		cout << "  ";
		for (int j = 0; j < tablero[0].size(); ++j) {
			switch (tablero[i][j]) {
			case 1:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+9*16);
				break;
			case 2:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+12*16);
				break;
			case 3:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+5*16); //NO HAY COLOR NARANJA
				break;
			case 4:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+2*16);
				break;
			case 5:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+6*16);
				break;
			case 6:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+14*16);
				break;
			case 7:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+10*16);
				break;
			case 8:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+10*16);
				break;
			case 9:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+11*16);
				break;
			default:SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15+13*16);
				break;
			}
			cout << " ";
			if (tablero[i][j] < 10) cout << " ";
			cout << tablero[i][j];
			cout << " ";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
		}
		cout << "\n";
	}
}

void nuevaPartida(vector<vector<int>>& tablero, int dificultad, int filas, int columnas, int puntuacion, int tam_tesela) {
	system("CLS");

	ShowWindow(GetConsoleWindow(), SW_MAXIMIZE);
	//IMPRIMIR TABLERO
	imprimirTablero(tablero);


	//LEER FILA Y COLUMNA DE LA JUGADA
	int fila = -1;
	bool quedanMovimientos = true;
	while (fila != 0 && quedanMovimientos) {
		printf("Puntuacion actual: %d,\n", puntuacion);
		while (fila < 0 || fila > filas) {
			cout << "Introduce la fila de la pieza a eliminar (0 para salir). Los rotores horizontales son:";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 10);
			cout << " 7";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
			cout << ", los verticales son: ";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 10);
			cout << " 8";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
			cout << ", las bombas TNT son: ";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 11);
			cout << " 9";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
			cout << ", las bombas puzzle son: ";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13);
			cout << "1X";
			SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
			cout << ", donde X indica el valor que eliminan.\n";
			while (!(cin >> fila)) {
				cin.clear();
				cin.ignore((std::numeric_limits<std::streamsize>::max)(), '\n');
				cerr << "Por favor, introduzca un numero de fila valido.\n";
			}
			if (fila < 0 || fila > filas) cout << "Fila fuera de rango.\n";
		}

		if (fila != 0) {
			int columna = 0;
			while (columna < 1 || columna > columnas) {
				cout << "Introduce la columna de la pieza a eliminar.\n";
				while (!(cin >> columna)) {
					cin.clear();
					cin.ignore((std::numeric_limits<std::streamsize>::max)(), '\n');
					cerr << "Por favor, introduzca un numero de columna valido.\n";
				}
				if (columna < 1 || columna > columnas) cout << "Columna fuera de rango.\n";
			}
			//FIN LEER FILA Y COLUMNA DE LA JUGADA

			int mayor = max(filas, columnas);

			//CUDA
			int *d_tablero;
			int *d_puntuacion;

			//DADO QUE CUDA NO SOPORTA VECTORES, PASAMOS EL VECTOR A ARRAY UNIDIMENSIONAL
			int* tablero_a = new int[tablero.size()*tablero[0].size()];
			for (int i = 0; i < tablero.size(); ++i) {
				for (int j = 0; j < tablero[0].size(); ++j) {
					tablero_a[i*tablero[0].size() + j] = tablero[i][j];
				}
			}

			//ALOCAMOS MEMORIA PARA EL TABLERO Y COPIAMOS NUESTRO ARRAY DE CPU A GPU
			hipMalloc(&d_tablero, (tablero.size()*tablero[0].size()*sizeof(int)));
			hipMemcpy(d_tablero, tablero_a, (tablero.size()*tablero[0].size()*sizeof(int)), hipMemcpyHostToDevice);

			hipMalloc(&d_puntuacion, (sizeof(int)));
			hipMemcpy(d_puntuacion, &puntuacion, sizeof(int), hipMemcpyHostToDevice);

			int n_bloques = (mayor+tam_tesela-1) / tam_tesela;

			dim3 DimGrid(n_bloques, n_bloques);
			dim3 DimBlock(tam_tesela, tam_tesela, 1);

			ToyBlast << < DimGrid, DimBlock >> > (d_tablero, filas, columnas, fila - 1, columna - 1, d_puntuacion);

			//UNA VEZ TERMINA, VOLVEMOS A COPIAR EL ARRAY DE GPU A CPU
			hipMemcpy(tablero_a, d_tablero, tablero.size()*tablero[0].size()*sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&puntuacion, d_puntuacion, sizeof(int), hipMemcpyDeviceToHost);


			//PASAMOS EL ARRAY A VECTOR
			for (int i = 0; i < tablero.size(); ++i) {
				for (int j = 0; j < tablero[0].size(); ++j) {
					tablero[i][j] = tablero_a[i*tablero[0].size() + j];
				}
			}

			//RELLENAMOS LOS CEROS CON ALEATORIO
			tableroAleatorio(tablero, dificultad);
			imprimirTablero(tablero);

			//LIBERAMOS MEMORIA DE GPU
			hipFree(d_tablero);
			hipFree(d_puntuacion);
			fila = -1;
			quedanMovimientos = quedanMovimientosF(tablero);
		}
		else {
			cout << "�Deseas guardar la partida? Introduzca 0 para no o 1 para si.\n";
			int guardar;
			while (!(cin >> guardar)) {
				cin.clear();
				cin.ignore((std::numeric_limits<std::streamsize>::max)(), '\n');
				cerr << "Por favor, introduzca un valor valido.\n";
			}
			if (guardar == 1) {
				string nombre;
				cout << "Introduzca el nombre de la partida a guardar.\n";
				cin >> nombre;
				guardarPartida(tablero, nombre, filas, columnas, dificultad, puntuacion);
			}
		}
	}
	if (!quedanMovimientos) {
		printf("No quedan movimientos posibles. Su puntuacion ha sido:\n %d \n GRACIAS POR JUGAR\n", puntuacion);
	}
}

void guardarPartida(vector<vector<int>> tablero, string nombre, int filas, int columnas, int dificultad, int puntuacion) {
	//GUARDAR LA PARTIDA SERIALIZANDO EL VECTOR COMO ARRAY UNIDIMENSIONAL DE ENTEROS
	ofstream outfile;
	outfile.open(nombre, ios::out | ios::trunc | ios::binary);

	outfile.write(reinterpret_cast<const char *>(&puntuacion), sizeof(int));
	outfile.write(reinterpret_cast<const char *>(&filas), sizeof(int));
	outfile.write(reinterpret_cast<const char *>(&dificultad), sizeof(int));
	for (int i = 0; i < tablero.size(); i++) {
		for (int j = 0; j < tablero[0].size(); j++) {
			outfile.write(reinterpret_cast<const char *>(&tablero[i][j]), sizeof(int));
			if (outfile.bad()) {
				throw std::runtime_error("Failed to write to outfile!");
			}
		}
	}
}

bool quedanMovimientosF(vector<vector<int>> tablero) {
	//SI NO HAY DOS PIEZAS JUNTAS EN NING�N ESPACIO DEL TABLERO, NO QUEDAN MOVIMIENTOS
	for (int i = 0; i < tablero.size(); i++) {
		for (int j = 0; j < tablero[0].size(); j++) {
			if (tablero[i][j]>6) return true;
			if ((i>0) && (tablero[i][j] == tablero[i - 1][j])) return true;
			if (((i + 1)<tablero.size()) && (tablero[i][j] == tablero[i + 1][j])) return true;
			if ((j>0) && (tablero[i][j] == tablero[i][j - 1])) return true;
			if (((j + 1)<tablero[0].size()) && (tablero[i][j] == tablero[i][j + 1])) return true;
		}
	}
	return false;
}


__global__ void ToyBlast(int *tablero, int filas, int columnas, int fila, int columna, int *puntuacion) {
	int hilo_fila = blockIdx.x*blockDim.x + threadIdx.x;
	int hilo_columna = blockIdx.y*blockDim.y + threadIdx.y;
	if (hilo_fila == fila && hilo_columna == columna) {
		int cont = 0;
		int valor = tablero[fila*columnas + columna];
		eliminarPieza(tablero, filas, columnas, fila, columna, valor, &cont);
		if (cont < 2) {
			tablero[fila*columnas + columna] = valor;
		}
		else {
			if (cont > 4) {
				//crearBomba
				switch (cont) {
				case 5: int aleatorio = clock() % 10;
					if (aleatorio < 5) {
						tablero[fila*columnas + columna] = 7; //SE CREA ALEATORIAMENTE UN ROTOR HORIZONTAL O VERTICAL
					}
					else {
						tablero[fila*columnas + columna] = 8;
					}
					break;
				case 6: tablero[fila*columnas + columna] = 9;
					break;
				default: tablero[fila*columnas + columna] = 10 + valor; //PARA ALMACENAR EL BLOQUE A EXPLOTAR DEL PUZZLE, LA BOMBA SERA DE 11 A 16 EN FUNCI�N DEL COLOR
					break;
				}
			}
		}

		if (cont>1) *puntuacion = *puntuacion + cont;
	}
	__syncthreads();
	
	
	//SUBIR TODOS LOS CEROS
	if (hilo_columna < columnas&&hilo_fila < filas) {
	for (int i = 1; i < filas; i++) {
		
			if (tablero[(filas - i)*columnas + hilo_columna] == 0) {
				if (tablero[(filas - (i + 1))*columnas + hilo_columna] == 0) {
					int j = i;
					while (tablero[(filas - (j + 1))*columnas + hilo_columna] == 0 && j < filas) {
						j++;
					}
					tablero[(filas - i)*columnas + hilo_columna] = tablero[(filas - (j + 1))*columnas + hilo_columna];
					tablero[(filas - (j + 1))*columnas + hilo_columna] = 0;
				}
				else {
					tablero[(filas - i)*columnas + hilo_columna] = tablero[(filas - (i + 1))*columnas + hilo_columna];
					tablero[(filas - (i + 1))*columnas + hilo_columna] = 0;
				}
			}
			__syncthreads();

		}
	}
}

__device__ void eliminarPieza(int *tablero, int filas, int columnas, int fila, int columna, int valor_ini, int *cont) {
	//DECLARAMOS BOOLEANOS PARA SABER HACIA DONDE TIENE QUE COMPROBAR SI TIENE QUE ELIMINAR LA PIEZA, SI TIENE QUE COMPROBARLA VOLVEMOS A LLAMAR A eliminarPieza
	int valor_act = tablero[fila*columnas + columna];

	if ((valor_act == valor_ini) && (valor_act<7)) {
		tablero[fila*columnas + columna] = 0;
		*cont = *cont + 1;

		bool arriba = true;
		bool abajo = true;
		bool izquierda = true;
		bool derecha = true;

		if (fila < 1) arriba = false;
		if (columna < 1) izquierda = false;
		if (fila + 1 >= filas) abajo = false;
		if (columna + 1 >= columnas) derecha = false;

		if (arriba) eliminarPieza(tablero, filas, columnas, fila - 1, columna, valor_ini, cont);
		if (izquierda) eliminarPieza(tablero, filas, columnas, fila, columna - 1, valor_ini, cont);
		if (abajo) eliminarPieza(tablero, filas, columnas, fila + 1, columna, valor_ini, cont);
		if (derecha) eliminarPieza(tablero, filas, columnas, fila, columna + 1, valor_ini, cont);
	}
	else {
		//SI NO COINICIDE EL VALOR, HAY QUE COMPROBAR SI ES UNA BOMBA, PERO S�LO HAY QUE EXPLOTARLA SI ES LA PRIMERA PIEZA ELIMINADA
		if ((*cont == 0) && (valor_act>6)) {
			//ES UNA BOMBA Y HAY QUE EXPLOTARLA
			*cont = 2;
			switch (valor_act) {
			case 7: //BOMBA 5 PIEZAS HORIZONTAL
				bombaRotorH(tablero, filas, columnas, fila, columna);
				break;
			case 8: //BOMBA 5 PIEZAS VERTICAL
				bombaRotorV(tablero, filas, columnas, fila, columna);
				break;
			case 9: //BOMBA 6 PIEZAS
				bombaTNT(tablero, filas, columnas, fila, columna);
				break;
			default://BOMBA 7 PIEZAS
				bombaPuzzle(tablero, filas, columnas, fila, columna);
				break;
			}
		}
		else {
			//ES UNA BOMBA PERO NO HAY QUE EXPLOTARLA, ES ADYACENTE A LAS QUE HAY QUE EXPLOTAR
		}
	}
}

__device__ void bombaRotorH(int* tablero, int filas, int columnas, int fila, int columna) {
	//BORRAR FILA
	for (int i = 0; i < columnas; i++) {
		tablero[fila*columnas + i] = 0;
	}
}

__device__ void bombaRotorV(int* tablero, int filas, int columnas, int fila, int columna) {
	//BORRAR COLUMNA
	for (int i = 0; i < filas; i++) {
		tablero[i*columnas + columna] = 0;
	}
}

__device__ void bombaTNT(int* tablero, int filas, int columnas, int fila, int columna) {
	tablero[fila*columnas + columna] = 0;
	bool arriba = true;
	bool abajo = true;
	bool izquierda = true;
	bool derecha = true;

	if (fila < 1) arriba = false;
	if (columna < 1) izquierda = false;
	if (fila + 1 >= filas) abajo = false;
	if (columna + 1 >= columnas) derecha = false;

	if (arriba) tablero[(fila - 1)*columnas + columna] = 0;
	if (izquierda) tablero[fila*columnas + (columna - 1)] = 0;
	if (abajo) tablero[(fila + 1)*columnas + columna] = 0;
	if (derecha) tablero[fila*columnas + (columna + 1)] = 0;
	if (arriba&&izquierda) tablero[(fila - 1)*columnas + (columna - 1)] = 0;
	if (arriba&&derecha) tablero[(fila - 1)*columnas + (columna + 1)] = 0;
	if (abajo&&izquierda) tablero[(fila + 1)*columnas + (columna - 1)] = 0;
	if (abajo&&derecha) tablero[(fila + 1)*columnas + (columna + 1)] = 0;
}

__device__ void bombaPuzzle(int* tablero, int filas, int columnas, int fila, int columna) {
	int valor = tablero[fila*columnas + columna] - 10;
	tablero[fila*columnas + columna] = 0;
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			if (tablero[i*columnas + j] == valor) tablero[i*columnas + j] = 0;
		}
	}
}